#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "gpuXmm.h"
#include "kernel.h"

#ifdef CUBLAS
void kernel_gpuXmm (hipblasHandle_t handle, unsigned int m, unsigned int n, unsigned int p, 
                    const gpuXmm_precision_t* a, const gpuXmm_precision_t* b, gpuXmm_precision_t* c)
{
    int size_a = m * n * sizeof(gpuXmm_precision_t);
    int size_b = n * p * sizeof(gpuXmm_precision_t);
    int size_c = m * p * sizeof(gpuXmm_precision_t);
    
    gpuXmm_precision_t* d_a;
    gpuXmm_precision_t* d_b;
    gpuXmm_precision_t* d_c;
    
	CHECK(hipMalloc(&d_a, size_a));
    CHECK(hipMalloc(&d_b, size_b));
    CHECK(hipMalloc(&d_c, size_c));

    CHECK(hipMemcpy(d_a, a, size_a, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_b, b, size_b, hipMemcpyHostToDevice));


    gpuXmm_precision_t alpha = 1.0f;
    gpuXmm_precision_t beta = 0.0f;
   
    #ifdef SP
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                p, m, n, &alpha, d_b, p, d_a, n, &beta, d_c, p);
    #else // DP
        hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                p, m, n, &alpha, d_b, p, d_a, n, &beta, d_c, p);
    #endif 
   
    CHECK(hipMemcpy(c, d_c, size_c, hipMemcpyDeviceToHost));

    CHECK(hipFree(d_a));
    CHECK(hipFree(d_b));
    CHECK(hipFree(d_c));
}
#endif


#ifdef CUBLAS_WO_DT
void kernel_gpuXmm (hipblasHandle_t handle, unsigned int m, unsigned int n, unsigned int p, 
                    const gpuXmm_precision_t* a, const gpuXmm_precision_t* b, gpuXmm_precision_t* c)
{ 
    gpuXmm_precision_t alpha = 1.0f;
    gpuXmm_precision_t beta = 0.0f;

    #ifdef SP
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                p, m, n, &alpha, b, p, a, n, &beta, c, p);
    #else // DP
        hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                p, m, n, &alpha, b, p, a, n, &beta, c, p);
    #endif
}
#endif
